#include <iostream>
#include <fstream>
#include <complex>
#include <chrono>
#include <cmath>
//#include <cuda.h>
#include "hip/hip_runtime.h"

// Ranges of the set
#define MIN_X -2
#define MAX_X 1
#define MIN_Y -1
#define MAX_Y 1

// Image ratio
//#define RATIO_X (MAX_X - MIN_X)
//#define RATIO_Y (MAX_Y - MIN_Y)

// Image size
#define RESOLUTION 1000
//#define WIDTH (RATIO_X * RESOLUTION)
//#define HEIGHT (RATIO_Y * RESOLUTION)

//#define STEP ((double)RATIO_X / WIDTH)

#define DEGREE 2        // Degree of the polynomial
#define ITERATIONS 1000 // Maximum number of iterations

using namespace std;



__global__ void mandelbrotGPUfunction(int *image, double step, double minX, double minY, int width, int height, int iterations)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos < width * height) {
        image[pos] = 0;

        const int row = pos / width;
        const int col = pos % width;

        const complex<double> c(col * step + minX, row * step + minY);

        complex<double> z(0, 0);

        for (int i = 1; i <= iterations; i++){
            z = pow(z, 2) + c;

            // If it is convergent
            if (abs(z) >= 2)
            {
                image[pos] = i;
                break;
            }
            if(i == iterations){
                image[pos]= 0;
            }
        }
    }
}



int main(int argc, char **argv)
{
    if (argc < 4) {
        cerr << "Usage: " << argv[0] << " <output file> <resolution> <block size>" << endl;
        return -1;
    }

    int resolution = atoi(argv[2]);
    int blockSize = atoi(argv[3]);

    // Adjust WIDTH and HEIGHT based on the resolution
    double ratioX = MAX_X - MIN_X;
    double ratioY = MAX_Y - MIN_Y;
    int WIDTH = ratioX * resolution;
    int HEIGHT = ratioY * resolution;
    double STEP = ((double)ratioX / WIDTH);

    int *const image = new int[HEIGHT * WIDTH];

    // Other code remains the same, adjust grid and block dimensions based on blockSize
    dim3 block(blockSize);
    dim3 grid((WIDTH * HEIGHT + block.x - 1) / block.x);


    // Timer
    hipEvent_t  start, stop;

    hipEventCreate( &start );
    hipEventCreate( &stop );

    // Allocation
    int *d_image;
    hipMalloc(&d_image, sizeof(int) * WIDTH * HEIGHT);

    // Start timer
    hipEventRecord(start);

    mandelbrotGPUfunction<<<grid, block>>>(d_image, STEP, MIN_X, MIN_Y, WIDTH, HEIGHT, ITERATIONS);
    hipDeviceSynchronize();
    hipMemcpy(image, d_image, sizeof(int) * WIDTH * HEIGHT, hipMemcpyDeviceToHost);

    // Stop timer
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsed_time = 0.0;
    hipEventElapsedTime(&elapsed_time, start, stop);

    cout << "Time elapsed: " <<  elapsed_time << " milliseconds." << endl;

    // Write the result to a file
    ofstream matrix_out;

    if (argc < 2)
    {
        cout << "Please specify the output file as a parameter." << endl;
        return -1;
    }

    matrix_out.open(argv[1], ios::trunc);
    if (!matrix_out.is_open())
    {
        cout << "Unable to open file." << endl;
        return -2;
    }

    for (int row = 0; row < HEIGHT; row++)
    {
        for (int col = 0; col < WIDTH; col++)
        {
            matrix_out << image[row * WIDTH + col];

            if (col < WIDTH - 1)
                matrix_out << ',';
        }
        if (row < HEIGHT - 1)
            matrix_out << endl;
    }
    matrix_out.close();

    delete[] image; // It's here for coding style, but useless
    hipFree(d_image);
    return 0;
}
